#include "hip/hip_runtime.h"
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include "TCVGeometry.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_fp16.h"
#include ""
//==========================================================================
// Kernels
//==========================================================================
__global__ void Resize_HF_Kernel(half * MemSrc, half * MemDst,int WidthSrc,int HeightSrc,int WidthDst,int HeightDst)
{
   //===============================================================================================
   //
   //===============================================================================================
   int globalX = blockIdx.x * blockDim.x + threadIdx.x;
   int globalY = blockIdx.y * blockDim.y + threadIdx.y;

   int OffsetMemDst = (globalY * WidthDst + globalX);

   float scaleWidth  =  (float)WidthDst / (float)(WidthSrc-1);
   float scaleHeight =  (float)HeightDst / (float)(HeightSrc-1);
   //===============================================================================================
   //
   //===============================================================================================
   if (globalX<WidthDst && globalY<HeightDst)
   {
        int x = (int)((float)globalX)/ scaleWidth;
        int y = (int)((float)globalY)/ scaleHeight;
        int x_1 = x-1;
        int y_1 = y-1;
        int x_2 = x+1;
        int y_2 = y+1;

        if(x_1 <= 0) x_1 = 0;
        if(y_1 <= 0) y_1 = 0;
        if(x_1 >= WidthSrc) x_1 = WidthSrc - 1;
        if(y_1 >= HeightSrc) y_1 = HeightSrc - 1;

        if(x_2 <= 0) x_2 = 0;
        if(y_2 <= 0) y_2 = 0;
        if(x_2 >= WidthSrc) x_2 = WidthSrc - 1;
        if(y_2 >= HeightSrc) y_2 = HeightSrc - 1;

		if(x <= 0) x = 0;
        if(y <= 0) y = 0;
        if(x >= WidthSrc) x = WidthSrc - 1;
        if(y >= HeightSrc) y = HeightSrc - 1;

        MemDst[OffsetMemDst] = __float2half( 0.25f*__half2float(MemSrc[y*WidthSrc+x]) + 0.125f*(__half2float(MemSrc[y*WidthSrc+x_1]) + __half2float(MemSrc[y*WidthSrc+x_2]) + __half2float(MemSrc[y_1*WidthSrc+x]) + __half2float(MemSrc[y_2*WidthSrc+x])) +
                          0.0625f*(__half2float(MemSrc[y_1*WidthSrc+x_1]) + __half2float(MemSrc[y_2*WidthSrc+x_1]) + __half2float(MemSrc[y_1*WidthSrc+x_2]) + __half2float(MemSrc[y_2*WidthSrc+x_2])));
   }
}
//--------------------------------------------------------------------------
__global__ void Resize_Kernel_Bilinear_HF(half* MemSrc, half * MemDst, int WidthSrc, int HeightSrc, int WidthDst, int HeightDst)
{
	//===============================================================================================
	//
	//===============================================================================================
	int globalX = blockIdx.x * blockDim.x + threadIdx.x;
	int globalY = blockIdx.y * blockDim.y + threadIdx.y;

	int OffsetMemDst = (globalY * WidthDst + globalX);
	//===============================================================================================
	//
	//===============================================================================================
	if (globalX < WidthDst && globalY < HeightDst)
	{
		float x_ratio = ((float)(WidthSrc - 1)) / (float)(WidthDst);
		float y_ratio = ((float)(HeightSrc - 1)) / (float)(HeightDst);

		int x = (int)(x_ratio * globalX);
		int y = (int)(y_ratio * globalY);

		float x_diff = (x_ratio * globalX) - x;
		float y_diff = (y_ratio * globalY) - y;
		int index = y * WidthSrc + x;

		// range is 0 to 255 thus bitwise AND with 0xff
		float A = __half2float(MemSrc[index]);
		float B = __half2float(MemSrc[index + 1]);
		float C = __half2float(MemSrc[index + WidthSrc]);
		float D = __half2float(MemSrc[index + WidthSrc + 1]);

		// Y = A(1-w)(1-h) + B(w)(1-h) + C(h)(1-w) + Dwh
		half Result = __float2half(A*(1.0f - x_diff)*(1.0f - y_diff) + B * (x_diff)*(1.0f - y_diff) +
					  C * (y_diff)*(1.0f - x_diff) + D * (x_diff*y_diff));

		MemDst[OffsetMemDst] = Result;
	}
}
template <typename T> __global__ void Resize_Kernel(T * MemSrc, T * MemDst, int WidthSrc, int HeightSrc, int WidthDst, int HeightDst)
{
	//===============================================================================================
	//
	//===============================================================================================
	int globalX = blockIdx.x * blockDim.x + threadIdx.x;
	int globalY = blockIdx.y * blockDim.y + threadIdx.y;

	int OffsetMemDst = (globalY * WidthDst + globalX);

	float scaleWidth = (float)(WidthDst)/(float)(WidthSrc-1);
	float scaleHeight = (float)(HeightDst)/(float)(HeightSrc-1);
	
	//===============================================================================================
	//
	//===============================================================================================
	if (globalX < WidthDst && globalY < HeightDst)
	{
		int x = (int)((float)globalX) / scaleWidth;
		int y = (int)((float)globalY) / scaleHeight;
		int x_1 = x - 1;
		int y_1 = y - 1;
		int x_2 = x + 1;
		int y_2 = y + 1;

		if (x_1 <= 0) x_1 = 0;
		if (y_1 <= 0) y_1 = 0;
		if (x_1 >= WidthSrc) x_1 = WidthSrc - 1;
		if (y_1 >= HeightSrc) y_1 = HeightSrc - 1;

		if (x_2 <= 0) x_2 = 0;
		if (y_2 <= 0) y_2 = 0;
		if (x_2 >= WidthSrc) x_2 = WidthSrc - 1;
		if (y_2 >= HeightSrc) y_2 = HeightSrc - 1;

		if (x >= WidthSrc)
		{
			printf("out X\n");
		}
		if (y >= HeightSrc)
		{
			printf("out Y\n");
		}

		if (x <= 0) x = 0;
		if (y <= 0) y = 0;
		if (x >= WidthSrc) x = WidthSrc - 1;
		if (y >= HeightSrc) y = HeightSrc - 1;


		T Result = (T)(0.25f*(MemSrc[y*WidthSrc + x]) + 0.125f*((MemSrc[y*WidthSrc + x_1]) + (MemSrc[y*WidthSrc + x_2]) + (MemSrc[y_1*WidthSrc + x]) + (MemSrc[y_2*WidthSrc + x])) +
			0.0625f*((MemSrc[y_1*WidthSrc + x_1]) + (MemSrc[y_2*WidthSrc + x_1]) + (MemSrc[y_1*WidthSrc + x_2]) + (MemSrc[y_2*WidthSrc + x_2])));

		if (Result > 255) Result = 255;

		MemDst[OffsetMemDst] = Result;
	}
}
//--------------------------------------------------------------------------
template <typename T>__global__ void Resize_Kernel_Bilinear(T* MemSrc, T * MemDst, int WidthSrc, int HeightSrc, int WidthDst, int HeightDst)
{
	//===============================================================================================
	//
	//===============================================================================================
	int globalX = blockIdx.x * blockDim.x + threadIdx.x;
	int globalY = blockIdx.y * blockDim.y + threadIdx.y;

	int OffsetMemDst = (globalY * WidthDst + globalX);
	//===============================================================================================
	//
	//===============================================================================================
	if (globalX < WidthDst && globalY < HeightDst)
	{
		float x_ratio = ((float)(WidthSrc - 1)) / (float)(WidthDst);
		float y_ratio = ((float)(HeightSrc - 1)) / (float)(HeightDst);

		int x = (int)(x_ratio * globalX);
		int y = (int)(y_ratio * globalY);

		float x_diff = (x_ratio * globalX) - x;
		float y_diff = (y_ratio * globalY) - y;
		int index = y*WidthSrc + x;

		// range is 0 to 255 thus bitwise AND with 0xff
		float A = (float)MemSrc[index];
		float B = (float)MemSrc[index + 1];
		float C = (float)MemSrc[index + WidthSrc];
		float D = (float)MemSrc[index + WidthSrc + 1];

		// Y = A(1-w)(1-h) + B(w)(1-h) + C(h)(1-w) + Dwh
		T Result = (T)(A*(1.0f - x_diff)*(1.0f - y_diff) + B*(x_diff)*(1.0f - y_diff) +
							 C*(y_diff)*(1.0f - x_diff) + D*(x_diff*y_diff));

		MemDst[OffsetMemDst] = Result;
	}
}
//--------------------------------------------------------------------------
__device__ int neumann_bc(int x, int nx)
{
    if(x < 0)
    {
        x = 0;
    }
    else if (x >= nx)
    {
        x = nx - 1;
    }
    return x;
}
//--------------------------------------------------------------------------
__global__ void Warping_HF_Kernel(half * MemSrc, half * MemDst,half * U,half * V,bool Inverted,int Width,int Height)
{
	   //===============================================================================================
	   //
	   //===============================================================================================
	   int globalX = blockIdx.x * blockDim.x + threadIdx.x;
	   int globalY = blockIdx.y * blockDim.y + threadIdx.y;

	   int GlobalOffset = (globalY * Width + globalX);
	   //===============================================================================================
	   //
	   //===============================================================================================
	   if (globalX<Width && globalY<Height)
	   {
		   float Result=0;
		   float uu=0,vv=0;
		   if (!Inverted)
		   {
		       uu =  (float)globalX-__half2float(U[GlobalOffset]);
			   vv =  (float)globalY-__half2float(V[GlobalOffset]);
		   }
		   else
		   {
			   uu =  (float)globalX+__half2float(U[GlobalOffset]);
			   vv =  (float)globalY+__half2float(V[GlobalOffset]);
		   }
		   //------------------------------------------
	       int sx = (uu < 0)? -1: 1;
	       int sy = (vv < 0)? -1: 1;

	       if (globalY==0) sy=1;

	       int x, y, dx, dy;

		   x  = neumann_bc((int) uu, Width);
	       y  = neumann_bc((int) vv, Height);
	       dx = neumann_bc((int) uu + sx, Width);
	       dy = neumann_bc((int) vv + sy, Height);

		   if (x>=0 && y>=0 && x<(Width) && y <(Height) && dx>=0 && dy >=0 && dx<(Width) && dy <(Height))
		   {
			  float p1 = __half2float(MemSrc[x  + Width * y]);
		      float p2 = __half2float(MemSrc[dx + Width * y]);
		      float p3 = __half2float(MemSrc[x  + Width * dy]);
			  float p4 = __half2float(MemSrc[dx + Width * dy]);

			  float e1 = ((float) sx * (uu - x));
			  float E1 = ((float) 1.0 - e1);
			  float e2 = ((float) sy * (vv - y));
			  float E2 = ((float) 1.0 - e2);

			 float w1 = E1 * p1 + e1 * p2;
		     float w2 = E1 * p3 + e1 * p4;

			 Result = E2 * w1 + e2 * w2;
		   }
		   //else Result = 0.0f;

		   MemDst[GlobalOffset]= __float2half(Result);
	   }
	   /*else
	   {
		   if (globalX>=0 && globalX<Width && globalY>=0 && globalY<Height)
			   MemDst[GlobalOffset]=__float2half_rn(0);
	   }*/
}
//--------------------------------------------------------------------------
__global__ void Warping_Uchar_Kernel(unsigned char * MemSrc, unsigned char * MemDst,half * U,half * V,bool Inverted,int Width,int Height)
{
	   //===============================================================================================
	   //
	   //===============================================================================================
	   int globalX = blockIdx.x * blockDim.x + threadIdx.x;
	   int globalY = blockIdx.y * blockDim.y + threadIdx.y;

	   int GlobalOffset = (globalY * Width + globalX);
	   //===============================================================================================
	   //
	   //===============================================================================================
	   if (globalX>=0 && globalX<Width && globalY>=0 && globalY<Height)
	   {
			  float Result;
	          //float uu = (float) (globalPosX + U[iGlobalOffset]);
	          //float vv = (float) (globalPosY + V[iGlobalOffset]);
			  float uu,vv;
			  if (Inverted==0)
			  {
				   uu = (float)globalX-__half2float(U[GlobalOffset]);
				   vv =  (float)globalY-__half2float(V[GlobalOffset]);
			  }
			  else
			  {
				   uu =  (float)globalX+__half2float(U[GlobalOffset]);
				   vv =  (float)globalY+__half2float(V[GlobalOffset]);
			  }

	          int sx = (uu < 0)? -1: 1;
	          int sy = (vv < 0)? -1: 1;

	          //printf("[%d]: shared value is %d\n", threadIdx.x, sx);
	          int x, y, dx, dy;

			  x  = neumann_bc((int) uu, Width);
	          y  = neumann_bc((int) vv, Height);
	          dx = neumann_bc((int) uu + sx, Width);
	          dy = neumann_bc((int) vv + sy, Height);

	          //dx = neumann_bc((int) ceil(uu), Width);
	          //dy = neumann_bc((int) ceil(vv), Height);

			  if (x>=0 && y>=0 && x<(Width) && y <(Height) && dx>=0 && dy >=0 && dx<(Width) && dy <(Height))
			  //{
			    //if ((x+Width*y)< TotalSize && (dx + Width * y)<TotalSize && (x  + Width * dy)<TotalSize && (dx + Width * dy)<TotalSize)
				{
				  float p1 =  (MemSrc[x  + Width * y]);
				  float p2 =  (MemSrc[dx + Width * y]);
				  float p3 =  (MemSrc[x  + Width * dy]);
				  float p4 =  (MemSrc[dx + Width * dy]);

				  float e1 = ((float) sx * (uu - x));
				  float E1 = ((float) 1.0 - e1);
				  float e2 = ((float) sy * (vv - y));
				  float E2 = ((float) 1.0 - e2);

				  float w1 = E1 * p1 + e1 * p2;
				  float w2 = E1 * p3 + e1 * p4;

				  Result = E2 * w1 + e2 * w2;
			  }
			  else Result = 0.0f;

			  if (Result>255) Result=255;

			  MemDst[GlobalOffset]=  (unsigned char)(Result);
	   }
}
//==========================================================================
// End Kernels
//==========================================================================
//--------------------------------------------------------------------------
TCVGeometry::TCVGeometry(void * d_Gpu)
{
    Gpu = d_Gpu;
}
//--------------------------------------------------------------------------
void TCVGeometry::Resize(TGpuMem::TGpuMemHalfFloat * MemSrc, TGpuMem::TGpuMemHalfFloat * MemDst)
{
	//----------------------------------------------------------------------------------------------------
	// Estimate the number of Blocks and number Threads
	//----------------------------------------------------------------------------------------------------
    dim3 numThreads = dim3(((TGpu *)Gpu)->GetBlockX(), ((TGpu *)Gpu)->GetBlockY(), 1);
    dim3 numBlocks =  dim3(((TGpu *)Gpu)->iDivUp(MemDst->Width(), numThreads.x), ((TGpu *)Gpu)->iDivUp(MemDst->Height(), numThreads.y));
	//----------------------------------------------------------------------------------------------------
	Resize_HF_Kernel<<<numBlocks, numThreads>>>(MemSrc->GetMemory(), MemDst->GetMemory(), MemSrc->Width(), MemSrc->Height(), MemDst->Width(), MemDst->Height());
    hipDeviceSynchronize();
}
//--------------------------------------------------------------------------
void TCVGeometry::ResizeBilinear(TGpuMem::TGpuMemHalfFloat * MemSrc, TGpuMem::TGpuMemHalfFloat * MemDst)
{
	//----------------------------------------------------------------------------------------------------
	// Estimate the number of Blocks and number Threads
	//----------------------------------------------------------------------------------------------------
	dim3 numThreads = dim3(((TGpu *)Gpu)->GetBlockX(), ((TGpu *)Gpu)->GetBlockY(), 1);
	dim3 numBlocks = dim3(((TGpu *)Gpu)->iDivUp(MemDst->Width(), numThreads.x), ((TGpu *)Gpu)->iDivUp(MemDst->Height(), numThreads.y));
	//----------------------------------------------------------------------------------------------------
	Resize_Kernel_Bilinear_HF << <numBlocks, numThreads >> > (MemSrc->GetMemory(), MemDst->GetMemory(), MemSrc->Width(), MemSrc->Height(), MemDst->Width(), MemDst->Height());
	hipDeviceSynchronize();
}
//--------------------------------------------------------------------------


void TCVGeometry::Resize(TGpuMem::TGpuMemUChar * MemSrc, TGpuMem::TGpuMemUChar * MemDst)
{
	//----------------------------------------------------------------------------------------------------
	// Estimate the number of Blocks and number Threads
	//----------------------------------------------------------------------------------------------------
    dim3 numThreads = dim3(((TGpu *)Gpu)->GetBlockX(), ((TGpu *)Gpu)->GetBlockY(), 1);
    dim3 numBlocks =  dim3(((TGpu *)Gpu)->iDivUp(MemDst->Width(), numThreads.x), ((TGpu *)Gpu)->iDivUp(MemDst->Height(), numThreads.y));
	//----------------------------------------------------------------------------------------------------
    Resize_Kernel<unsigned char><<<numBlocks, numThreads>>>(MemSrc->GetMemory(), MemDst->GetMemory(), MemSrc->Width(), MemSrc->Height(), MemDst->Width(), MemDst->Height());
    hipDeviceSynchronize();
}
//--------------------------------------------------------------------------
void TCVGeometry::ResizeBilinear(TGpuMem::TGpuMemUChar * MemSrc, TGpuMem::TGpuMemUChar * MemDst)
{
	//----------------------------------------------------------------------------------------------------
	// Estimate the number of Blocks and number Threads
	//----------------------------------------------------------------------------------------------------
	dim3 numThreads = dim3(((TGpu *)Gpu)->GetBlockX(), ((TGpu *)Gpu)->GetBlockY(), 1);
	dim3 numBlocks = dim3(((TGpu *)Gpu)->iDivUp(MemDst->Width(), numThreads.x), ((TGpu *)Gpu)->iDivUp(MemDst->Height(), numThreads.y));
	//----------------------------------------------------------------------------------------------------
	Resize_Kernel_Bilinear<unsigned char> << <numBlocks, numThreads >> > (MemSrc->GetMemory(), MemDst->GetMemory(), MemSrc->Width(), MemSrc->Height(), MemDst->Width(), MemDst->Height());
	hipDeviceSynchronize();
}
//--------------------------------------------------------------------------
void TCVGeometry::Warping(TGpuMem::TGpuMemHalfFloat * MemSrc, TGpuMem::TGpuMemHalfFloat * MemDst, TGpuMem::TGpuMemHalfFloat * MemU, TGpuMem::TGpuMemHalfFloat * MemV,bool Inverted)
{
	//----------------------------------------------------------------------------------------------------
	// Estimate the number of Blocks and number Threads
	//----------------------------------------------------------------------------------------------------
    dim3 numThreads = dim3(((TGpu *)Gpu)->GetBlockX(), ((TGpu *)Gpu)->GetBlockY(), 1);
    dim3 numBlocks =  dim3(((TGpu *)Gpu)->iDivUp(MemSrc->Width(), numThreads.x), ((TGpu *)Gpu)->iDivUp(MemSrc->Height(), numThreads.y));
	//----------------------------------------------------------------------------------------------------
    Warping_HF_Kernel<<<numBlocks, numThreads>>>(MemSrc->GetMemory(), MemDst->GetMemory(), MemU->GetMemory(), MemV->GetMemory(), Inverted, MemSrc->Width(), MemSrc->Height());
    hipDeviceSynchronize();
}
//--------------------------------------------------------------------------
void TCVGeometry::Warping(TGpuMem::TGpuMemUChar * MemSrc, TGpuMem::TGpuMemUChar * MemDst, TGpuMem::TGpuMemHalfFloat * MemU, TGpuMem::TGpuMemHalfFloat * MemV,bool Inverted)
{
	//----------------------------------------------------------------------------------------------------
	// Estimate the number of Blocks and number Threads
	//----------------------------------------------------------------------------------------------------
    dim3 numThreads = dim3(((TGpu *)Gpu)->GetBlockX(), ((TGpu *)Gpu)->GetBlockY(), 1);
    dim3 numBlocks =  dim3(((TGpu *)Gpu)->iDivUp(MemDst->Width(), numThreads.x), ((TGpu *)Gpu)->iDivUp(MemDst->Height(), numThreads.y));
	//----------------------------------------------------------------------------------------------------
    Warping_Uchar_Kernel<<<numBlocks, numThreads>>>(MemSrc->GetMemory(), MemDst->GetMemory(), MemU->GetMemory(), MemV->GetMemory(), Inverted, MemSrc->Width(), MemSrc->Height());
    hipDeviceSynchronize();
}
//--------------------------------------------------------------------------
TCVGeometry::~TCVGeometry()
{
	//delete CV;
}
//--------------------------------------------------------------------------

